
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdint>
#include <memory>
#include <algorithm>
#include <chrono>

#include <hiprand/hiprand.h>
#include <hipblas.h>

const float ALPHA{ 2.0f };
const float BETA{ 30.0f };

// Must be multiples of 16 for wmma code to work
const int32_t MATRIX_M{ 1024 };
const int32_t MATRIX_N{ 1024 };
const int32_t MATRIX_K{ 1024 };

__global__ void convertFp32ToFp16(float* in, half* out, const int32_t len) {
    int32_t idx{ static_cast<int32_t>(blockIdx.x * blockDim.x + threadIdx.x) };
    if (idx < len) {
        out[idx] = static_cast<half>(in[idx]);
    }
}

__global__ void convertFp16ToFp32(half* in, float* out, const int32_t len) {
    int32_t idx{ static_cast<int32_t>(blockIdx.x * blockDim.x + threadIdx.x) };
    if (idx < len) {
        out[idx] = static_cast<float>(in[idx]);
    }
}

void referenceMatrixMultiply(
    float* a,
    float* b,
    float* bias,
    float* c_out,
    int32_t M,
    int32_t N,
    int32_t K,
    float alpha,
    float beta) {
    // Note: All input/output matrices are in column-major memory order
    for (int32_t aRow{ 0 }; aRow < M; aRow++) {
        for (int32_t bCol{ 0 }; bCol < N; bCol++) {
            // Compute the value at `aRow, bCol`
            for (int32_t k{ 0 }; k < K; k++) {
                c_out[aRow + bCol * M] += a[aRow + k * M] * b[k + bCol * K];
            }

            // Scale the `a * b` result by `alpha`
            c_out[aRow + bCol * M] *= alpha;

            // Add in the `bias` scaled by `beta`
            c_out[aRow + bCol * M] += beta * bias[aRow + bCol * M];
        }
    }
}

int32_t main() {
    float* a_fp32;
    float* b_fp32;
    float* bias_fp32;
    float* c_fp32;

    half* a_fp16;
    half* b_fp16;
    half* bias_fp16;
    half* c_fp16;

    // Allocate the memory for the matrices
    hipMalloc(&a_fp32, MATRIX_M * MATRIX_K * sizeof(float));
    hipMalloc(&b_fp32, MATRIX_K * MATRIX_N * sizeof(float));
    hipMalloc(&bias_fp32, MATRIX_M * MATRIX_N * sizeof(float));
    hipMalloc(&c_fp32, MATRIX_M * MATRIX_N * sizeof(float));
    hipMalloc(&a_fp16, MATRIX_M * MATRIX_K * sizeof(half));
    hipMalloc(&b_fp16, MATRIX_K * MATRIX_N * sizeof(half));
    hipMalloc(&bias_fp16, MATRIX_M * MATRIX_N * sizeof(half));
    hipMalloc(&c_fp16, MATRIX_M * MATRIX_N * sizeof(half));

    std::unique_ptr<float[]> a_host{ new float[MATRIX_M * MATRIX_K * sizeof(float)] };
    std::unique_ptr<float[]> b_host{ new float[MATRIX_K * MATRIX_N * sizeof(float)] };
    std::unique_ptr<float[]> bias_host{ new float[MATRIX_M * MATRIX_N * sizeof(float)] };
    std::unique_ptr<float[]> c_host{ new float[MATRIX_M * MATRIX_N * sizeof(float)] };
    std::unique_ptr<float[]> c_cublas_host{ new float[MATRIX_M * MATRIX_N * sizeof(float)] };

    // Clear the contents of `c` matrices
    hipMemset(c_fp16, __float2half(0.0f), MATRIX_M * MATRIX_N * sizeof(half));
    std::fill(c_host.get(), c_host.get() + MATRIX_M * MATRIX_N, 0.0f);

    // Initialize the cuBLAS handle to use tensor cores
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);

    // Create and initialize the CUDA random number generator
    hiprandGenerator_t randGen;
    hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(randGen, 69);

    // Create and initialize the CUDA events
    hipEvent_t startCublas, stopCublas;
    hipEventCreate(&startCublas);
    hipEventCreate(&stopCublas);

    // Create the timing variables for the reference CPU matrix multiply
    std::chrono::high_resolution_clock::time_point referenceStartTime, referenceEndTime;

    // Curand does not support `half`, so generate random `float` and then convert to `half`
    {
        hiprandGenerateUniform(randGen, a_fp32, MATRIX_M * MATRIX_K);
        hiprandGenerateUniform(randGen, b_fp32, MATRIX_K * MATRIX_N);
        hiprandGenerateUniform(randGen, bias_fp32, MATRIX_M * MATRIX_N);

        const int32_t BLOCK_SIZE{ 256 };
        dim3 gridDimA{ (MATRIX_M * MATRIX_K + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 gridDimB{ (MATRIX_K * MATRIX_N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 gridDimBias{ (MATRIX_M * MATRIX_N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 blockDim{ BLOCK_SIZE, 1, 1 };

        // Create and initialize the CUDA streams
        hipStream_t stream1, stream2, stream3;

        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);

        // Launch both kernels to convert the `a_fp32` and `b_fp32`
        convertFp32ToFp16<<<gridDimA, blockDim, 0, stream1>>>(a_fp32, a_fp16, MATRIX_M * MATRIX_K);
        convertFp32ToFp16<<<gridDimB, blockDim, 0, stream2>>>(b_fp32, b_fp16, MATRIX_K * MATRIX_N);
        convertFp32ToFp16<<<gridDimBias, blockDim, 0, stream3>>>(
            bias_fp32, bias_fp16, MATRIX_M * MATRIX_N);

        // Wait for both streams
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);

        // Clean up
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);
    }

    // Copy the random constents of the device float matrices to the host matrices
    hipMemcpy(
        a_host.get(), a_fp32, MATRIX_M * MATRIX_K * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(
        b_host.get(), b_fp32, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(
        bias_host.get(), bias_fp32, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost);

    // Some useful prints
    std::cout << "M = " << MATRIX_M << ", "
              << "N = " << MATRIX_N << ", "
              << "K = " << MATRIX_K << ", "
              << std::fixed << std::setprecision(2)
              << "alpha = " << ALPHA << ", "
              << "beta = " << BETA << std::endl;

    std::cout << "Running with cuBLAS" << std::endl;

    // Perform the cuBLAS matrix multiplication
    {
        // Warm up cuBLAS
        hipblasGemmEx(
            cublasHandle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            MATRIX_M,
            MATRIX_N,
            MATRIX_K,
            &ALPHA,
            a_fp16,
            HIP_R_16F,
            MATRIX_M,
            b_fp16,
            HIP_R_16F,
            MATRIX_K,
            &ALPHA,
            c_fp16,
            HIP_R_16F,
            MATRIX_M,
            HIPBLAS_COMPUTE_32F,
            CUBLAS_GEMM_DEFAULT_TENSOR_OP);

        hipblasAxpyEx(
            cublasHandle,
            MATRIX_M * MATRIX_N,
            &BETA,
            HIP_R_32F,
            bias_fp16,
            HIP_R_16F,
            1,
            c_fp16,
            HIP_R_16F,
            1,
            HIP_R_32F);

        // Reset the contents of `c_fp16` matrices
        hipMemset(c_fp16, __float2half(0.0f), MATRIX_M * MATRIX_N * sizeof(half));

        // Launch the cuBLAS matrix multiplication kernel
        hipEventRecord(startCublas);
        hipblasGemmEx(
            cublasHandle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            MATRIX_M,
            MATRIX_N,
            MATRIX_K,
            &ALPHA,
            a_fp16,
            HIP_R_16F,
            MATRIX_M,
            b_fp16,
            HIP_R_16F,
            MATRIX_K,
            &BETA,
            c_fp16,
            HIP_R_16F,
            MATRIX_M,
            HIPBLAS_COMPUTE_32F,
            CUBLAS_GEMM_DEFAULT_TENSOR_OP);

        // Add the bias in to the result. Treat the `bias_fp16` and `c_fp16`
        // as long vectors when adding them together
        hipblasAxpyEx(
            cublasHandle,
            MATRIX_M * MATRIX_N,
            &BETA,
            HIP_R_32F,
            bias_fp16,
            HIP_R_16F,
            1,
            c_fp16,
            HIP_R_16F,
            1,
            HIP_R_32F);
        hipEventRecord(stopCublas);
        hipEventSynchronize(stopCublas);

        // Convert the `c_fp16` to `c_fp32`
        dim3 blockDimConvert{ 256, 1, 1 };
        dim3 gridDimConvert{
            (MATRIX_M * MATRIX_N + blockDimConvert.x - 1) / blockDimConvert.x, 1, 1 };

        // Launch the kernel to convert the `c_fp16`
        convertFp16ToFp32<<<gridDimConvert, blockDimConvert>>>(c_fp16, c_fp32, MATRIX_M * MATRIX_N);

        // Copy the result to the `c_cublas_host`
        hipMemcpy(
            c_cublas_host.get(), c_fp32, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost);
    }

    std::cout << "Running on CPU" << std::endl;

    // Perform the CPU matrix multiplication
    {
        referenceStartTime = std::chrono::high_resolution_clock::now();
        referenceMatrixMultiply(
            a_host.get(),
            b_host.get(),
            bias_host.get(),
            c_host.get(),
            MATRIX_M,
            MATRIX_N,
            MATRIX_K,
            ALPHA,
            BETA);
        referenceEndTime = std::chrono::high_resolution_clock::now();
    }

    // Compare the matrix outputs
    {
        // Usa a 1% relative tolerance
        int32_t errors = 0;
        for (int32_t i = 0; i < MATRIX_M * MATRIX_N; i++) {
            float v1 = c_host[i];
            float v2 = c_cublas_host[i];
            float diff  = fabs(v1 - v2);
            float relative_err = diff / v2;
            float eps = 0.01;
            if (relative_err >= eps) {
                errors++;
                if (errors < 10) {
                    std::cout << v1 << " " << v2 << std::endl;
                }
            }
        }

        if (errors > 0) {
            std::cout << "cuBLAS does not agree with reference! " << errors
                      << " errors!" << std::endl;
        } else {
            std::cout << "Results verified: reference and cuBLAS agree." << std::endl;
            float cublasTime;
            hipEventElapsedTime(&cublasTime, startCublas, stopCublas);

            std::chrono::milliseconds referenceDuration{
                std::chrono::duration_cast<std::chrono::milliseconds>(
                    referenceEndTime - referenceStartTime) };
            std::cout << std::fixed << std::setprecision(4)
                      << "cuBLAS took " << cublasTime << "ms\n"
                      << "reference took " << referenceDuration.count() << "ms" << std::endl;
        }
    }

    // Clean up
    hiprandDestroyGenerator(randGen);
    hipblasDestroy(cublasHandle);
    hipEventDestroy(startCublas);
    hipEventDestroy(stopCublas);

    // Free all of the allocated memory
    hipFree(a_fp32);
    hipFree(b_fp32);
    hipFree(bias_fp32);
    hipFree(c_fp32);
    hipFree(a_fp16);
    hipFree(b_fp16);
    hipFree(bias_fp16);
    hipFree(c_fp16);

    return 0;
}
