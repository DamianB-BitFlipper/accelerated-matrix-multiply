
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdint>

#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <mma.h>

using namespace nvcuda;

const int32_t WARP_SIZE{ 32 };

// Must be multiples of 16 for wmma code to work
const int32_t MATRIX_M{ 16384 };
const int32_t MATRIX_N{ 16384 };
const int32_t MATRIX_K{ 16384 };

// The only dimensions currently supported by WMMA
const int32_t WMMA_M{ 16 };
const int32_t WMMA_N{ 16 };
const int32_t WMMA_K{ 16 };

__global__ void wmma_matmul(
    half* a,
    half* b,
    half* bias,
    half* c_out,
    int32_t M,
    int32_t N,
    int32_t K,
    float alpha,
    float beta) {
    // Convert the `alpha` and `beta` weights into half precision
    const half alpha_fp16{ __float2half(alpha) };
    const half beta_fp16{ __float2half(beta) };

    // Tile using a 2D grid
    int32_t warpI = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int32_t warpJ = blockIdx.y * blockDim.y + threadIdx.y;

    // Define the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> bias_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    const int32_t aRow{ WMMA_M * warpJ };
    const int32_t bCol{ WMMA_N * warpI };

    for (int32_t i{ 0 }; i < K; i += WMMA_K) {
        const int32_t aCol{ i };
        const int32_t bRow{ i };

        // Bounds checking
        if (aRow < M && bCol < N) {
            // Fill the fragments noting the column-major memory format
            wmma::load_matrix_sync(a_frag, a + aCol * M + aRow, M);
            wmma::load_matrix_sync(b_frag, b + bCol * K + bRow, K);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Load and add in the bias
    wmma::load_matrix_sync(bias_frag, bias + bCol * M + aRow, M, wmma::mem_col_major);

#pragma unroll
    for (int32_t i{ 0 }; i < acc_frag.num_elements; i++) {
        c_frag.x[i] = alpha_fp16 * acc_frag.x[i] + beta_fp16 * bias_frag.x[i];
    }

    // Store the resulting output
    wmma::store_matrix_sync(c_out + bCol * M + aRow, c_frag, M, wmma::mem_col_major);
}

__global__ void convertFp32ToFp16(float* in, half* out, const int32_t len) {
    int32_t idx{ static_cast<int32_t>(blockIdx.x * blockDim.x + threadIdx.x) };
    if (idx < len) {
        out[idx] = static_cast<half>(in[idx]);
    }
}

int32_t main() {
    float* a_fp32;
    float* b_fp32;
    float* bias_fp32;

    half* a_fp16;
    half* b_fp16;
    half* bias_fp16;
    half* c_fp16;

    const float alpha{ 2.0f };
    const float beta{ 2.0f };

    // Create and initialize the CUDA random number generator
    hiprandGenerator_t randGen;
    hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(randGen, 69);

    // Create and initialize the CUDA events
    hipEvent_t startWMMA, stopWMMA;
    hipEventCreate(&startWMMA);
    hipEventCreate(&stopWMMA);

    // Allocate the memory for the matrices
    hipMalloc(&a_fp32, MATRIX_M * MATRIX_K * sizeof(float));
    hipMalloc(&b_fp32, MATRIX_K * MATRIX_N * sizeof(float));
    hipMalloc(&bias_fp32, MATRIX_M * MATRIX_N * sizeof(float));
    hipMalloc(&a_fp16, MATRIX_M * MATRIX_K * sizeof(half));
    hipMalloc(&b_fp16, MATRIX_K * MATRIX_N * sizeof(half));
    hipMalloc(&bias_fp16, MATRIX_M * MATRIX_N * sizeof(half));
    hipMalloc(&c_fp16, MATRIX_M * MATRIX_N * sizeof(half));

    // Clear the contents of `c_fp16`
    hipMemset(c_fp16, __float2half(0.0f), MATRIX_M * MATRIX_N * sizeof(half));

    // Curand does not support `half`, so generate random `float` and then convert to `half`
    {
        hiprandGenerateUniform(randGen, a_fp32, MATRIX_M * MATRIX_K);
        hiprandGenerateUniform(randGen, b_fp32, MATRIX_K * MATRIX_N);
        hiprandGenerateUniform(randGen, bias_fp32, MATRIX_K * MATRIX_N);

        const int32_t BLOCK_SIZE{ 256 };
        dim3 gridDimA{ (MATRIX_M * MATRIX_K + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 gridDimB{ (MATRIX_K * MATRIX_N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 gridDimBias{ (MATRIX_M * MATRIX_N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 blockDim{ BLOCK_SIZE, 1, 1 };

        // Create and initialize the CUDA streams
        hipStream_t stream1, stream2, stream3;

        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);

        // Launch both kernels to convert the `a_fp32` and `b_fp32`
        convertFp32ToFp16<<<gridDimA, blockDim, 0, stream1>>>(a_fp32, a_fp16, MATRIX_M * MATRIX_K);
        convertFp32ToFp16<<<gridDimB, blockDim, 0, stream2>>>(b_fp32, b_fp16, MATRIX_K * MATRIX_N);
        convertFp32ToFp16<<<gridDimBias, blockDim, 0, stream3>>>(
            bias_fp32, bias_fp16, MATRIX_M * MATRIX_N);

        // Wait for both streams
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);

        // Clean up
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);
    }

    // Some useful prints
    std::cout << "M = " << MATRIX_M << ", ";
    std::cout << "N = " << MATRIX_N << ", ";
    std::cout << "K = " << MATRIX_K << ", ";
    std::cout << std::fixed << std::setprecision(2) << "alpha = " << alpha << ", ";
    std::cout << std::fixed << std::setprecision(2) << "beta = " << beta << std::endl;

    std::cout << "Running with WMMA" << std::endl;

    // Perform the WMMA matrix multiplication
    {
        dim3 blockDim{ 256, 4, 1 };
        dim3 gridDim{
            (MATRIX_N + (WMMA_N * blockDim.x / WARP_SIZE) - 1) / (WMMA_N * blockDim.x / WARP_SIZE),
            (MATRIX_M + (WMMA_M * blockDim.y) - 1) / (WMMA_M * blockDim.y),
            1 };

        // Launch the WMMA matrix multiplication kernel
        hipEventRecord(startWMMA);
        wmma_matmul<<<gridDim, blockDim>>>(
            a_fp16,
            b_fp16,
            bias_fp16,
            c_fp16,
            MATRIX_M,
            MATRIX_N,
            MATRIX_K,
            alpha,
            beta);
        hipEventRecord(stopWMMA);
        hipEventSynchronize(stopWMMA);
    }

    // Clean up
    hiprandDestroyGenerator(randGen);
    hipEventDestroy(startWMMA);
    hipEventDestroy(stopWMMA);

    // Free all of the allocated memory
    hipFree(a_fp32);
    hipFree(b_fp32);
    hipFree(bias_fp32);
    hipFree(a_fp16);
    hipFree(b_fp16);
    hipFree(bias_fp16);
    hipFree(c_fp16);

    return 0;
}
