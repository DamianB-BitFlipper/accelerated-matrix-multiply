
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>

#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <mma.h>

using namespace nvcuda;

const int32_t WARP_SIZE{ 32 };

// Must be multiples of 16 for wmma code to work
const int32_t MATRIX_M{ 16384 };
const int32_t MATRIX_N{ 16384 };
const int32_t MATRIX_K{ 16384 };

// The only dimensions currently supported by WMMA
const int32_t WMMA_M{ 16 };
const int32_t WMMA_N{ 16 };
const int32_t WMMA_K{ 16 };

__global__ void wmma_matmul(
    half* a,
    half* b,
    half* c,
    int32_t M,
    int32_t N,
    int32_t K,
    float alpha,
    float beta) {

    // Tile using a 2D grid
    int32_t warpI = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
    int32_t warpJ = blockIdx.y * blockDim.y + threadIdx.y;

    // Define the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);
}

__global__ void convertFp32ToFp16(float* in, half* out, const int32_t len) {
    int32_t idx{ blockIdx.x * blockDim.x + threadIdx.x };
    if (idx < len) {
        out[idx] = static_cast<half>(in[idx]);
    }
}

int32_t main() {
    float* a_fp32;
    float* b_fp32;
    float* bias_fp32;

    half* a_fp16;
    half* b_fp16;
    half* bias_fp16;
    half* c_fp16;

    // Create and initialize the CUDA random number generator
    hiprandGenerator_t randGen;
    hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(randGen, 69);

    // Create and initialize the CUDA events
    hipEvent_t startWMMA, stopWMMA;
    hipEventCreate(&startWMMA);
    hipEventCreate(&stopWMMA);

    // Allocate the memory for the matrices
    hipMalloc(&a_fp32, MATRIX_M * MATRIX_K * sizeof(float));
    hipMalloc(&b_fp32, MATRIX_K * MATRIX_N * sizeof(float));
    hipMalloc(&bias_fp32, MATRIX_M * MATRIX_N * sizeof(float));
    hipMalloc(&a_fp16, MATRIX_M * MATRIX_K * sizeof(half));
    hipMalloc(&b_fp16, MATRIX_K * MATRIX_N * sizeof(half));
    hipMalloc(&bias_fp16, MATRIX_M * MATRIX_N * sizeof(half));
    hipMalloc(&c_fp16, MATRIX_M * MATRIX_N * sizeof(half));

    // Clear the contents of `c_fp16`
    hipMemset(c_fp16, __float2half(0.0f), MATRIX_M * MATRIX_N * sizeof(half));

    // Curand does not support `half`, so generate random `float` and then convert to `half`
    {
        hiprandGenerateUniform(randGen, a_fp32, MATRIX_M * MATRIX_K);
        hiprandGenerateUniform(randGen, b_fp32, MATRIX_K * MATRIX_N);
        hiprandGenerateUniform(randGen, bias_fp32, MATRIX_K * MATRIX_N);

        const int32_t BLOCK_SIZE{ 256 };
        dim3 gridDimA{ (MATRIX_M * MATRIX_K + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 gridDimB{ (MATRIX_K * MATRIX_N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 gridDimBias{ (MATRIX_M * MATRIX_N + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };
        dim3 blockDim{ BLOCK_SIZE, 1, 1 };

        // Create and initialize the CUDA streams
        hipStream_t stream1, stream2, stream3;

        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);

        // Launch both kernels to convert the `a_fp32` and `b_fp32`
        convertFp32ToFp16<<<gridDimA, blockDim, 0, stream1>>>(a_fp32, a_fp16, MATRIX_M * MATRIX_K);
        convertFp32ToFp16<<<gridDimB, blockDim, 0, stream2>>>(b_fp32, b_fp16, MATRIX_K * MATRIX_N);
        convertFp32ToFp16<<<gridDimBias, blockDim, 0, stream3>>>(
            bias_fp32, bias_fp16, MATRIX_M * MATRIX_N);

        // Wait for both streams
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);
        hipStreamSynchronize(stream3);

        // Clean up
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);
    }

    // Clean up
    hiprandDestroyGenerator(randGen);
    hipEventDestroy(startWMMA);
    hipEventDestroy(stopWMMA);

    hipFree(a_fp32);
    hipFree(b_fp32);
    hipFree(bias_fp32);
    hipFree(a_fp16);
    hipFree(b_fp16);
    hipFree(bias_fp16);
    hipFree(c_fp16);

    return 0;
}
